#include "hip/hip_runtime.h"
/*----------------------------------------------
create by Ding Ze An
Date : 2023 5 23
National Taitung Unv. IPGIT
Email : andy856996@gamil.com
----------------------------------------------*/
#include <omp.h>
#include <stdio.h>
#include <string.h>
#include "mex.h"

void convolution2D(double *mask, double *kernel, double *output, int m_row, int m_col, int k_row, int k_col) {
    int i, j, m, n;
    int o_row = m_row + k_row - 1;
    int o_col = m_col + k_col - 1;
    for (i = 0; i < o_row; i++) {
        for (j = 0; j < o_col; j++) {
            double temp = 0.0;
            for (m = 0; m < k_row; m++) {
                for (n = 0; n < k_col; n++) {
                    int mask_row = i - m;
                    int mask_col = j - n;

                    if (mask_row >= 0 && mask_row < m_row && mask_col >= 0 && mask_col < m_col) {
                        int mask_index = mask_row * m_col + mask_col;
                        int kernel_index = m * k_col + n;
                        temp += mask[mask_index] * kernel[kernel_index];
                    }
                }
            }
            output[i * o_col + j] = temp;
        }
    }
}
__global__ static void convolution2D_CUDA
(double *mask, double *kernel, double *output, int m_row, int m_col, int k_row, int k_col){
	int m, n;
	const int j = blockDim.x * blockIdx.x+threadIdx.x;
	const int i = blockDim.y * blockIdx.y+threadIdx.y;
    int o_row = m_row + k_row - 1;
    int o_col = m_col + k_col - 1;
	if(i<o_row&&j<o_col){
		double temp = 0.0;
		for (m = 0; m < k_row; m++) {
			for (n = 0; n < k_col; n++) {
				int mask_row = i - m;
				int mask_col = j - n;
				if (mask_row >= 0 && mask_row < m_row && mask_col >= 0 && mask_col < m_col) {
					int mask_index = mask_row * m_col + mask_col;
					int kernel_index = m * k_col + n;
					temp += mask[mask_index] * kernel[kernel_index];
				}
			}
		}
		output[i * o_col + j] = temp;
	}
}
__global__ static void GPUArr2MatlabArr(double* output ,double* input,int R,int C){
	const int j = blockDim.x * blockIdx.x+threadIdx.x;
	const int i = blockDim.y * blockIdx.y+threadIdx.y;
	if(i < row && j < col){
		output[j * o_row + i] = input[i * o_col + j];
	}
}
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
    double *mask;
    double *kernel;
    double *output;
    int mrows, mcols, krows, kcols;

    mask = mxGetPr(prhs[0]);
    kernel = mxGetPr(prhs[1]);

    mrows = mxGetM(prhs[0]);
    mcols = mxGetN(prhs[0]);

    krows = mxGetM(prhs[1]);
    kcols = mxGetN(prhs[1]);
	
	int output_dim_R = mrows + krows - 1;
	int output_dim_C = mcols + kcols - 1;
	
    plhs[0] = mxCreateDoubleMatrix(mrows + krows - 1, mcols + kcols - 1, mxREAL);
    output = mxGetPr(plhs[0]);
	
	hipMalloc((void**)&mask_gpu, mrows * mcols * sizeof(double));
	hipMalloc((void**)&kernal_gpu, krows * kcols* sizeof(double));
	hipMalloc((void**)&output_gpu, output_dim_R * output_dim_C * sizeof(double));
	hipMemcpy(mask_gpu, mask, mrows * mcols * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(kernal_gpu, kernal, krows * kcols* sizeof(double), hipMemcpyHostToDevice);
	
	const int threadNum = 32;
	dim3 threads(threadNum, threadNum, 1); 
	dim3 grid((output_dim_C + threads.x - 1) / threads.x, (output_dim_R + threads.y - 1) / threads.y, 1);
	convolution2D_CUDA<<<grid,threads>>>(mask_gpu, kernal_gpu, output_gpu, mrows, mcols, krows, kcols);
	
	hipFree(mask_gpu);
	hipFree(kernal_gpu);
	
	hipMalloc((void**)&outputFinal_gpu, output_dim_R * output_dim_C * sizeof(double));
	GPUArr2MatlabArr<<<grid,threads>>>(outputFinal_gpu,output_gpu,R,C);
	hipFree(output_gpu);
	
	hipMemcpy(output, outputFinal_gpu,output_dim_R * output_dim_C * sizeof(double), hipMemcpyDeviceToHost);
	
    hipFree(outputFinal_gpu);
}