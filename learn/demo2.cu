#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void add(int *a, int *b, int *c, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

void random_ints(int* a, int n)
{
    for (int i = 0; i < n; ++i)
        a[i] = rand();
}

int main()
{
    int n = (20480 * 20480);
    int threads_per_block = 512;
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    int size = n * sizeof(int);
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int*)malloc(size);
    random_ints(a, n);
    b = (int*)malloc(size);
    random_ints(b, n);
    c = (int*)malloc(size);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<(n + threads_per_block - 1)/threads_per_block, threads_per_block>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    // for (int i = 0; i < n; ++i) {
    //     cout << c[i] << ",";
    // }
    cout << endl;
    cout << hipGetErrorString(hipGetLastError()) << endl;

    free(a);
    free(b);
    free(c);

    return 0;
}